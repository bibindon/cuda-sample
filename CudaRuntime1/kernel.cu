#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <chrono>

#define N 1024  // 行列サイズ

// CUDAカーネル（GPUで並列処理）
__global__ void matrixMulCUDA(float* A, float* B, float* C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        float sum = 0.0f;
        for (int i = 0; i < n; i++) {
            sum += A[row * n + i] * B[i * n + col];
        }
        C[row * n + col] = sum;
    }
}

// CPU版の行列乗算
void matrixMulCPU(float* A, float* B, float* C, int n) {
    for (int row = 0; row < n; row++) {
        for (int col = 0; col < n; col++) {
            float sum = 0.0f;
            for (int i = 0; i < n; i++) {
                sum += A[row * n + i] * B[i * n + col];
            }
            C[row * n + col] = sum;
        }
    }
}

int main() {
    size_t bytes = N * N * sizeof(float);

    // ホストメモリ確保（CPU側）
    float* h_A = (float*)malloc(bytes);
    float* h_B = (float*)malloc(bytes);
    float* h_C = (float*)malloc(bytes);
    float* h_C_GPU = (float*)malloc(bytes);

    // 行列の初期化（ランダム値）
    for (int i = 0; i < N * N; i++) {
        h_A[i] = rand() % 100 / 100.0f;
        h_B[i] = rand() % 100 / 100.0f;
    }

    // **CPU実行**
    auto start_cpu = std::chrono::high_resolution_clock::now();
    matrixMulCPU(h_A, h_B, h_C, N);
    auto end_cpu = std::chrono::high_resolution_clock::now();
    float time_cpu = std::chrono::duration<float, std::milli>(end_cpu - start_cpu).count();
    printf("CPU Execution Time: %f ms\n", time_cpu);

    // **GPU実行**
    float* d_A, * d_B, * d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid(N / threadsPerBlock.x, N / threadsPerBlock.y);

    auto start_gpu = std::chrono::high_resolution_clock::now();
    matrixMulCUDA << <blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C, N);
    hipDeviceSynchronize();
    auto end_gpu = std::chrono::high_resolution_clock::now();
    float time_gpu = std::chrono::duration<float, std::milli>(end_gpu - start_gpu).count();

    printf("GPU Execution Time: %f ms\n", time_gpu);

    // 結果取得
    hipMemcpy(h_C_GPU, d_C, bytes, hipMemcpyDeviceToHost);

    // メモリ解放
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(h_A); free(h_B); free(h_C); free(h_C_GPU);

    return 0;
}

/*
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
*/
